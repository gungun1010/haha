#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__global__ void gpu_matrixmult(float *a, float *b, float *c, int n, int m, int p) {

  int col = threadIdx.x + blockDim.x * blockIdx.x;
  int row = threadIdx.y + blockDim.y * blockIdx.y;

  int indexb = col;
  int index = row * n + col;
  
  if(col < m && row < n) {
    c[index] = 0.;
    for (int indexa = row*n; indexa < (row*n + p); indexa++, indexb+=m){ 
        if(threadIdx.x == 0 && threadIdx.y == 0)
      printf("a = %.2f (indexa = %d), b = %.2f (indexb = %d)\n", a[indexa], indexa, b[indexb], indexb);
      c[index] += a[indexa]*b[indexb];
    }
  }

}


void cpu_matrixmult(float *a,float *b, float *c, int n) {

  int index, indexa, indexb;
  float cvalue;
  for(int col=0;col < n; col++)
    for(int row=0;row < n; row++) {
      indexb = col;
      index = row * n + col;
      cvalue = 0.;
      for (indexa = row*n; indexa < (row*n + n); indexa++, indexb+=n) 
	cvalue += a[indexa]*b[indexb];
      c[index] -= cvalue; //NOTE: This calculates the diff between CPU and GPU computations.
    }
}


int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int Grid_Dim = 1; //Grid dimension, x and y, square
  int Block_Dim = 1; //Block dimension, x and y, square

  int n,m,p; // matrix dimension
  float *a,*b,*c;
  float *dev_a, *dev_b, *dev_c;
  int size_a, size_b, size_c; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else printf("Device count = %d\n",gpucount);

  if (argc<4) {
    printf("Usage: Task1GPUsp <n> <m> <p> <block dim> <grid dim>\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  m = atoi(argv[2]);
  p = atoi(argv[3]);
  

  Block_Dim = atoi(argv[4]); // non-Square block
  if (Block_Dim * Block_Dim > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  Grid_Dim = atoi(argv[5]); // Square grid
  if (Grid_Dim*Block_Dim < n || Grid_Dim*Block_Dim < m) {
    printf("Error, number of threads in x/y dimensions less than number of array elements\n");
    exit (-1);
  }
  
  printf("A Matrix Dimension = %dx%d\n",n,p);
  printf("B Matrix Dimension = %dx%d\n",p,m);
  printf("C Matrix Dimension = %dx%d\n",n,m);
  printf("Block_Dim = %d, Grid_Dim = %d\n",Block_Dim,Grid_Dim);

  dim3 Grid(Grid_Dim, Grid_Dim); //Grid structure
  dim3 Block(Block_Dim, Block_Dim); //Block structure

  size_a = n * p * sizeof(float); // number of bytes in total in arrays
  size_b = p * m * sizeof(float); // number of bytes in total in arrays
  size_c = n * m * sizeof(float); // number of bytes in total in arrays

  a = (float*) malloc(size_a); // dynamically allocated memory for arrays on host
  b = (float*) malloc(size_b);
  c = (float*) malloc(size_c); // results from GPU

  srand(12345);
  //int p = n; //Used here only to illustrate proper initialization for non-square case
  printf ("a\n");
  for(i=0;i < n;i++){
    for(j=0;j < p;j++) {
      // a[i * n + j] = (float) rand() / (float) RAND_MAX;
      a[i * p + j] = (float) (i+j);
      printf("%.2f  ", a[i * n + j]);
    }
    printf("\n");
  }

  printf("b\n");
  for(i=0;i < p;i++){
    for(j=0;j < m;j++) {
      //b[i * n + j] = (float) rand() / (float) RAND_MAX;
      b[i * m + j] = (float) (i+j);
      printf("%.2f  ", b[i * n + j]);
    }
    printf("\n");
  }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, size_a); // allocate memory on device
  hipMalloc((void**)&dev_b, size_b);
  hipMalloc((void**)&dev_c, size_c);

  hipMemcpy(dev_a, a , size_a ,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , size_b ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed

  gpu_matrixmult<<<Grid,Block>>>(dev_a,dev_b,dev_c,n,m,p);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  hipMemcpy(c,dev_c, size_c ,hipMemcpyDeviceToHost);

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time
  printf("c\n");
  for(i=0;i < n;i++){
    for(j=0;j < m;j++) {
      printf("%.2f  ", c[i * n + j]);
    }
    printf("\n");
  }
  exit(1);
  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

  hipEventRecord(start, 0); // use same timing
  // cudaEventSynchronize(start); // not needed

  cpu_matrixmult(a,b,c, n); // do calculation on host (NOTE: This computes the diff with GPU result.)

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------

  double error, suma, sumb, sumc, ai, bi, ci;
  suma = 0.; sumb = 0; sumc = 0;
  for(i=0;i < n*n;i++) {
    ai = (double) a[i];
    bi = (double) b[i];
    ci = (double) c[i];
    suma += ai*ai;
    sumb += bi*bi;
    sumc += ci*ci;
  }
  suma = sqrt(suma);
  sumb = sqrt(sumb);
  sumc = sqrt(sumc);
  error =  sumc/(n*suma*sumb);
  printf("Scaled error between GPU and CPU: %e\n", error);

// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}