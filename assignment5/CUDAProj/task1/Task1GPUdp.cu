#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__global__ void gpu_matrixmult(double *a, double *b, double *c, int n, int m, int p) {

  int col = threadIdx.x + blockDim.x * blockIdx.x;
  int row = threadIdx.y + blockDim.y * blockIdx.y;

  int indexb = col;
  int index = row * m + col;

  if(col < m && row < n) {
    c[index] = 0.;
    for (int indexa = row*p; indexa < (row*p + p); indexa++, indexb+=m){ 
      c[index] += a[indexa]*b[indexb];
    }
  }
}


void cpu_matrixmult(double *a,double *b, double *c, int n, int m, int p) {

  int index, indexa, indexb;
  double cvalue;
  for(int col=0;col < m; col++)
    for(int row=0;row < n; row++) {
      indexb = col;
      index = row * m + col;
      cvalue = 0.;
      for (indexa = row*p; indexa < (row*p + p); indexa++, indexb+=m) 
	cvalue += a[indexa]*b[indexb];
      c[index] -= cvalue; //NOTE: This calculates the diff between CPU and GPU computations.
    }
}


int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int Grid_Dim_x = 1; //Grid dimension, x
  int Grid_Dim_y = 1; //Grid dimension, y
  int Block_Dim_x = 1; //Block dimension, x
  int Block_Dim_y = 1; //Block dimension, y

  int n,m,p; // matrix dimension
  double *a,*b,*c;
  double *dev_a, *dev_b, *dev_c;
  int size_a, size_b, size_c; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  //else printf("Device count = %d\n",gpucount);
  if (sizeof(argv)<8) {
    printf("Usage: Task1GPUsp <n> <m> <p> <block dim x> <block dim y> <grid dim x> <grid dim y>\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  m = atoi(argv[2]);
  p = atoi(argv[3]);
  

  Block_Dim_x = atoi(argv[4]); // non-Square block, x dimension size (# of cols)
  Block_Dim_y = atoi(argv[5]); // non-Square block, y dimension size (# of rows)
  if (Block_Dim_x * Block_Dim_y > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  Grid_Dim_x = atoi(argv[6]); // non-Square grid, x diemnsion size (# of cols)
  Grid_Dim_y = atoi(argv[7]); // non-Square grid, y dimension size (# of rows)
  if (Grid_Dim_x*Block_Dim_x < m ) {
    printf("Error, number of threads in x dimensions less than number of array elements\n");
    exit (-1);
  }

  if (Grid_Dim_y*Block_Dim_y < n) {
    printf("Error, number of threads in y dimensions less than number of array elements\n");
    exit (-1);
  }
  
  //printf("A Matrix Dimension = %dx%d\n",n,p);
  //printf("B Matrix Dimension = %dx%d\n",p,m);
  //printf("C Matrix Dimension = %dx%d\n",n,m);
  //printf("Block_x = %d Block_y = %d, Grid_x = %d Grid_y = %d\n",Block_Dim_x, Block_Dim_y,Grid_Dim_x, Grid_Dim_y);

  dim3 Grid(Grid_Dim_x, Grid_Dim_y); //Grid structure
  dim3 Block(Block_Dim_x, Block_Dim_y); //Block structure

  size_a = n * p * sizeof(double); // number of bytes in total in arrays
  size_b = p * m * sizeof(double); // number of bytes in total in arrays
  size_c = n * m * sizeof(double); // number of bytes in total in arrays

  a = (double*) malloc(size_a); // dynamically allocated memory for arrays on host
  b = (double*) malloc(size_b);
  c = (double*) malloc(size_c); // results from GPU

  srand(12345);
  //int p = n; //Used here only to illustrate proper initialization for non-square case
  
  //printf ("a\n");
  for(i=0;i < n;i++){
    for(j=0;j < p;j++) {
      a[i * p + j] = (double) rand() / (double) RAND_MAX;
      //a[i * p + j] = (double) (i+j);
      //printf("%.2f  ", a[i * p + j]);
    }
    //printf("\n");
  }

  //printf("b\n");
  for(i=0;i < p;i++){
    for(j=0;j < m;j++) {
      b[i * m + j] = (double) rand() / (double) RAND_MAX;
      //b[i * m + j] = (double) (i+j);
      //printf("%.2f  ", b[i * m + j]);
    }
    //printf("\n");
  }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, size_a); // allocate memory on device
  hipMalloc((void**)&dev_b, size_b);
  hipMalloc((void**)&dev_c, size_c);

  hipMemcpy(dev_a, a , size_a ,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , size_b ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed

  gpu_matrixmult<<<Grid,Block>>>(dev_a,dev_b,dev_c,n,m,p);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  hipMemcpy(c,dev_c, size_c ,hipMemcpyDeviceToHost);

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time
  /*
  printf("c\n");
  for(i=0;i < n;i++){
    for(j=0;j < m;j++) {
      printf("%.2f  ", c[i * m + j]);
    }
    printf("\n");
  }*/
  
  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)
/*
  cudaEventRecord(start, 0); // use same timing
  // cudaEventSynchronize(start); // not needed

  cpu_matrixmult(a,b,c, n, m, p); // do calculation on host (NOTE: This computes the diff with GPU result.)

  cudaEventRecord(stop, 0); // instrument code to measue end time
  cudaEventSynchronize(stop);
  cudaEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------

  double error, suma, sumb, sumc, ai, bi, ci;
  suma = 0.; sumb = 0; sumc = 0;
  for(i=0;i < n*n;i++) {
    ai = (double) a[i];
    bi = (double) b[i];
    ci = (double) c[i];
    suma += ai*ai;
    sumb += bi*bi;
    sumc += ci*ci;
  }
  suma = sqrt(suma);
  sumb = sqrt(sumb);
  sumc = sqrt(sumc);
  error =  sumc/(n*suma*sumb);
  printf("Scaled error between GPU and CPU: %e\n", error);
  
*/
// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
