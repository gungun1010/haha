#include "hip/hip_runtime.h"
/*
 *  Compilation: gcc -Wall ex1.c -o ex1 -L/home/leon/CUDAProj/finalProj/clamav/lib -lclamav
*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include </home/leon/clamav/include/clamav.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TAG_BITS 10

const char *BYTE_CODE = "bytecode.cvd";
const char *DAILY = "/dailyPack/dailyGPUsig.bin";
const char *MAIN = "/mainPack/mainGPUsig.bin";

__global__ void patternMatching(char *set1, char *set2, char *set3){
}

//function to construct the signature database for GPU
void loadSig (const char *fileName, char **buffer, size_t *size){
    long lSize;
    FILE *fp;
    char *fullPath; 
    
    fullPath = (char *)malloc(100*sizeof(char));

    //find file
    strcpy(fullPath, (char *) cl_retdbdir());//this function returns where the signature file is
    strcat(fullPath, "/");
    strcat(fullPath, fileName);
    fp = fopen (fullPath , "rb" );
    if( !fp ) 
        perror(fileName),exit(1);

    //seek the end of file
    fseek( fp , 0L , SEEK_END);
    lSize = ftell( fp );
    rewind( fp );
    //printf("%ld\n",lSize);
    (*size) = lSize;

    /* allocate memory for entire content */
    (*buffer) = (char *) calloc( 1, lSize+1 );
    if( !(*buffer) ) 
        fclose(fp),fputs("memory alloc fails",stderr),exit(1);

    /* copy the file into the buffer */
    if( 1!=fread( (*buffer) , lSize, 1 , fp) )
        fclose(fp),free((*buffer)),fputs("entire read fails",stderr),exit(1);

    fclose(fp);
    //free(buffer);
}

//function to load input file to scan
void loadFile (const char *fileName, char **buffer, size_t *size){
    long lSize;
    FILE *fp;
    char *fullPath; 
    
    fullPath = (char *)malloc(100*sizeof(char));
    //find file
    strcpy(fullPath, (char *) cl_retdbdir());//this function returns where the signature file is
    //printf("%s\n",fullPath);
    strcat(fullPath, "/");
    //printf("%s\n",fullPath);
    strcat(fullPath, fileName);
    //printf("find signature file in %s\n",fullPath);
    fp = fopen (fullPath , "rb" );
    if( !fp ) perror(fileName),exit(1);
    
    //seek the beginning of file
    //fseek(fp, SEEK_SET, 0);
    fseek( fp , 0L , SEEK_END);
    lSize = ftell( fp );
    rewind( fp );
    //printf("%ld\n",lSize);
    (*size) = lSize;

    /* allocate memory for entire content */
    (*buffer) = (char *) calloc( 1, lSize+1 );
    if( !(*buffer) ) 
        fclose(fp),fputs("memory alloc fails",stderr),exit(1);

    /* copy the file into the buffer */
    if( 1!=fread( (*buffer) , lSize, 1 , fp) )
          fclose(fp),free((*buffer)),fputs("entire read fails",stderr),exit(1);

    fclose(fp);
    //free(buffer);
}
/*
 * Exit codes:
 *  0: clean
 *  1: infected
 *  2: error
 */
//const char *DBDIR = "/home/leon/clamav/share/clamav";

int main(int argc, char **argv)
{
	int fd, ret;
	unsigned long int size = 0;
	unsigned int sigs = 0;
	long double mb;
	const char *virname;
	struct cl_engine *engine;

    int gpucount = 0; // Count of available GPUs
    //We only have 3701312 signatures
    //each thread get 1 signature, we need no more than 1024*1024 threads
    //grid size is then fixed to (32,32,1), and block size is (32,32,1)

    int Grid_Dim_x = 256; //Grid dimension, x
    int Grid_Dim_y = 256; //Grid dimension, y
    int Block_Dim_x = 32; //Block dimension, x
    int Block_Dim_y = 32; //Block dimension, y
    hipError_t errorcode;

    //host buffer to store each signature dataset
    char *byteCodeBuf; 
    char *dailyBuf;
    char *mainBuf;
    char *devBcb, *devDb, *devMb;//device buffer correspoding to the host buffer
    size_t sizeBcb, sizeDb, sizeMb;
   
    // --------------------SET PARAMETERS AND DATA -----------------------
    //load signatures into host buffer
    //loadFile(BYTE_CODE, &byteCodeBuf, &sizeBcb);
    loadSig(DAILY, &dailyBuf, &sizeDb);
    loadSig(MAIN, &mainBuf, &sizeMb);
    
    for(int i=0; i<11; i++){
        printf("%x ", (unsigned char) dailyBuf[i]);
    }

    exit(1);
    //loadFile(MAIN, &mainBuf, &sizeMb);

    errorcode = hipGetDeviceCount(&gpucount);
    if (errorcode == hipErrorNoDevice) {
        printf("No GPUs are visible\n");
        exit(-1);
    }

    if (Block_Dim_x * Block_Dim_y > 1024) {
        printf("Error, too many threads in block\n");
        exit (-1);
    }

    dim3 Grid(Grid_Dim_x, Grid_Dim_y); //Grid structure
    dim3 Block(Block_Dim_x, Block_Dim_y); //Block structure
    
    hipMalloc((void**)&devBcb, sizeBcb*sizeof(char));
    hipMalloc((void**)&devDb, sizeDb*sizeof(char));
    hipMalloc((void**)&devMb, sizeMb*sizeof(char)); 

    hipMemcpy(devBcb, byteCodeBuf , sizeBcb ,hipMemcpyHostToDevice);
    hipMemcpy(devDb, dailyBuf , sizeDb ,hipMemcpyHostToDevice);
    hipMemcpy(devMb, mainBuf , sizeMb ,hipMemcpyHostToDevice);

    if(argc != 2) {
        printf("Usage: %s file\n", argv[0]);
        return 2;
    }

    if((fd = open(argv[1], O_RDONLY)) == -1) {
        printf("Can't open file %s\n", argv[1]);
        return 2;
    }


    if((ret = cl_init(CL_INIT_DEFAULT)) != CL_SUCCESS) {
        printf("Can't initialize libclamav: %s\n", cl_strerror(ret));
        return 2;
    }

    if(!(engine = cl_engine_new())) {
        printf("Can't create new engine\n");
        return 2;
    }
    /* load all available databases from default directory */
    printf("loading signatures in %s\n",cl_retdbdir());
    if((ret = cl_load(cl_retdbdir(), engine, &sigs, CL_DB_STDOPT)) != CL_SUCCESS) {
        printf("cl_load: %s\n", cl_strerror(ret));
        close(fd);
            cl_engine_free(engine);
        return 2;
    }

    printf("Loaded %u signatures.\n", sigs);

    /* build engine */
    if((ret = cl_engine_compile(engine)) != CL_SUCCESS) {
        printf("Database initialization error: %s\n", cl_strerror(ret));;
            cl_engine_free(engine);
        close(fd);
        return 2;
    }

    /* scan file descriptor */
    if((ret =cl_scandesc(fd, &virname, &size, engine, CL_SCAN_STDOPT)) == CL_VIRUS) {
        printf("Virus detected: %s\n", virname);
    } else {
        if(ret == CL_CLEAN) {
            printf("No virus detected.\n");
        } else {
            printf("Error: %s\n", cl_strerror(ret));
            cl_engine_free(engine);
            close(fd);
            return 2;
        }
    }
    close(fd);

    /* free memory */
    cl_engine_free(engine);

    /* calculate size of scanned data */
    mb = size * (CL_COUNT_PRECISION / 1024) / 1024.0;
    printf("Data scanned:%ld  %2.2Lf MB\n", size, mb);

    return ret == CL_VIRUS ? 1 : 0;
}
