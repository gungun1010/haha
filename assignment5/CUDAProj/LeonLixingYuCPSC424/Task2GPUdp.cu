#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>


__global__ void matmul_tile(double *a, double *b, double *c, int n, int m, int p, int TW) {
  extern __shared__ double bigarray[]; 

  double *aTile=&bigarray[0], *bTile=&bigarray[TW*TW];
  int tx = threadIdx.x; 
  int ty = threadIdx.y; 
  double cvalue = 0; 
  int col = tx + blockDim.x * blockIdx.x;
  int row = ty + blockDim.y * blockIdx.y;
  int tileNum, aIdx, bIdx;
   
   tileNum = p/TW + (p % TW != 0);

  for (int tileIdx=0; tileIdx<tileNum; tileIdx++) {
    aIdx = tileIdx*TW + tx;
    if(aIdx >= p || row >= n){
        aTile[ty*TW+tx] = 0.;
    }else{
        aTile[ty*TW+tx] = a[row*p + aIdx]; //Copy to shared memory 
    }
    
    bIdx = tileIdx*TW +ty;
    if(bIdx >= p || col >= m){
        bTile[ty*TW+tx] = 0.;
    }else{
        bTile[ty*TW+tx] = b[bIdx*m + col]; //Copy to shared memory 
    }

    __syncthreads();
    for (int k=0; k<TW; k++){
         cvalue += aTile[ty*TW+k] * bTile[k*TW+tx];
         //printf("bx = %d, by = %d, tx = %d, ty = %d: a=%.2f b=%.2f\n",blockIdx.x, blockIdx.y, tx, ty, aTile[ty*TW+k],bTile[k*TW+tx]);
    }
    __syncthreads();
    
  }
  
  if(row < n && col <m){
      c[row*m + col] = cvalue;
  }
}


void cpu_matrixmult(double *a,double *b, double *c, int n, int m, int p) {

  int index, indexa, indexb;
  double cvalue;
  for(int col=0;col < m; col++)
    for(int row=0;row < n; row++) {
      indexb = col;
      index = row * m + col;
      cvalue = 0.;
      for (indexa = row*p; indexa < (row*p + p); indexa++, indexb+=m) 
	cvalue += a[indexa]*b[indexb];
      c[index] -= cvalue; //NOTE: This calculates the diff between CPU and GPU computations.
    }
}


int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int Grid_Dim_x = 1; //Grid dimension, x
  int Grid_Dim_y = 1; //Grid dimension, y
  int Block_Dim_x = 1; //Block dimension, x
  int Block_Dim_y = 1; //Block dimension, y
  int TW = 1;

  int n,m,p; // matrix dimension
  double *a,*b,*c;
  double *dev_a, *dev_b, *dev_c;
  int size_a, size_b, size_c; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  //else printf("Device count = %d\n",gpucount);
  if (argc<9) {
    printf("# of inputs: %d\n", argc);
    printf("Usage: Task1GPUsp <n> <m> <p> <block dim x> <block dim y> <grid dim x> <grid dim y> <tile width>\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  m = atoi(argv[2]);
  p = atoi(argv[3]);
  

  Block_Dim_x = atoi(argv[4]); // non-Square block, # of rows
  Block_Dim_y = atoi(argv[5]); // non-Square block, # of cols
  if (Block_Dim_x * Block_Dim_y > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  //not really used in Task2 
  Grid_Dim_x = atoi(argv[6]); // non-Square grid, # of rows
  Grid_Dim_y = atoi(argv[7]); // non-Square grid, # of cols
  
  TW = atoi(argv[8]);
   
  if(Block_Dim_x != Block_Dim_y || Block_Dim_x != TW || Block_Dim_y != TW){
      printf("Error, bx, by, tw must be equal\n");
      exit(-1);
  }

  //printf("A Matrix Dimension = %dx%d\n",n,p);
  //printf("B Matrix Dimension = %dx%d\n",p,m);
  //printf("C Matrix Dimension = %dx%d\n",n,m);
  Grid_Dim_x = m/Block_Dim_x + (m % Block_Dim_x != 0);
  Grid_Dim_y = n/Block_Dim_y + (n % Block_Dim_y != 0);
  //printf("Grid_x = %d Grid_y = %d\n", Grid_Dim_x,Grid_Dim_y);

  dim3 Grid(Grid_Dim_x, Grid_Dim_y); //Grid structure
  dim3 Block(Block_Dim_x, Block_Dim_y); //Block structure

  size_a = n * p * sizeof(double); // number of bytes in total in arrays
  size_b = p * m * sizeof(double); // number of bytes in total in arrays
  size_c = n * m * sizeof(double); // number of bytes in total in arrays

  a = (double*) malloc(size_a); // dynamically allocated memory for arrays on host
  b = (double*) malloc(size_b);
  c = (double*) malloc(size_c); // results from GPU

  srand(12345);
  //int p = n; //Used here only to illustrate proper initialization for non-square case
  
  //printf ("a\n");
  for(i=0;i < n;i++){
    for(j=0;j < p;j++) {
      a[i * p + j] = (double) rand() / (double) RAND_MAX;
      //a[i * p + j] = (double) (i+j);
      //printf("%.2f  ", a[i * p + j]);
    }
    //printf("\n");
  }

  //printf("b\n");
  for(i=0;i < p;i++){
    for(j=0;j < m;j++) {
      b[i * m + j] = (double) rand() / (double) RAND_MAX;
      //b[i * m + j] = (double) (i+j);
      //printf("%.2f  ", b[i * m + j]);
    }
    //printf("\n");
  }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, size_a); // allocate memory on device
  hipMalloc((void**)&dev_b, size_b);
  hipMalloc((void**)&dev_c, size_c);

  hipMemcpy(dev_a, a , size_a ,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , size_b ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed
  size_t Ns = 2 * TW*TW * sizeof(double);
  matmul_tile<<<Grid,Block, Ns>>>(dev_a,dev_b,dev_c,n,m,p,TW);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  hipMemcpy(c,dev_c, size_c ,hipMemcpyDeviceToHost);

  //printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time
  
  //printf("c\n");
  for(i=0;i < n;i++){
    for(j=0;j < m;j++) {
      printf("%.2f  ", c[i * m + j]);
    }
    printf("\n");
  }
  
  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)
/*
  cudaEventRecord(start, 0); // use same timing
  // cudaEventSynchronize(start); // not needed

  cpu_matrixmult(a,b,c, n,m,p); // do calculation on host (NOTE: This computes the diff with GPU result.)

  cudaEventRecord(stop, 0); // instrument code to measue end time
  cudaEventSynchronize(stop);
  cudaEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------

  double error, suma, sumb, sumc, ai, bi, ci;
  suma = 0.; sumb = 0; sumc = 0;
  for(i=0;i < n*n;i++) {
    ai = (double) a[i];
    bi = (double) b[i];
    ci = (double) c[i];
    suma += ai*ai;
    sumb += bi*bi;
    sumc += ci*ci;
  }
  suma = sqrt(suma);
  sumb = sqrt(sumb);
  sumc = sqrt(sumc);
  error =  sumc/(n*suma*sumb);
  printf("Scaled error between GPU and CPU: %e\n", error);
  
*/
// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
