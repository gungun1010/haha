#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__global__ void matmul_tile(float *a, float *b, float *c, int n, int m, int p, int TW, int NTB) {
  extern __shared__ float bigarray[]; 

  float *aTile=&bigarray[0], *bTile=&bigarray[TW*TW];
  int tx = threadIdx.x; 
  int ty = threadIdx.y; 
  float *cvalue;//scope: thread 
  int col = tx + blockDim.x * blockIdx.x;
  int row = ty + blockDim.y * blockIdx.y;
  int tileNum, aIdx, bIdx, tileIdx_m, tileCol;

  tileNum = p/TW + (p % TW != 0);

  cvalue = (float *)malloc(NTB*sizeof(float));

  //init c tiles
  for (tileIdx_m=0; tileIdx_m<NTB; tileIdx_m++) cvalue[tileIdx_m] = 0.;

  for (int tileIdx_p=0; tileIdx_p<tileNum; tileIdx_p++) {
    
    //load aTile
    aIdx = tileIdx_p*TW + tx;
    if(aIdx >= p || row >= n){
        aTile[ty*TW+tx] = 0.;
    }else{
        aTile[ty*TW+tx] = a[row*p + aIdx]; //Copy to shared memory 
    }
    
    for(tileIdx_m=0; tileIdx_m<NTB; tileIdx_m++){ 
        //load btile[ty][tx] with element [ty][tx] in tileIdx_m-th tile of b
        if((blockIdx.x % NTB) == 0){
            bIdx = tileIdx_p*TW +ty;
            tileCol = tx + blockDim.x*(blockIdx.x+tileIdx_m);
            if(bIdx >= p || tileCol >= m){
                bTile[ty*TW+tx] = 0.;
            }else{
                bTile[ty*TW+tx] = b[bIdx*m + tileCol]; //Copy to shared memory 
            }

            __syncthreads();
            for (int k=0; k<TW; k++){
                 cvalue[tileIdx_m] += aTile[ty*TW+k] * bTile[k*TW+tx];
                 //printf("bx = %d, by = %d, (tx = %d, ty = %d) @ tileIdx_m = %d : a=%.2f b=%.2f \n",blockIdx.x, blockIdx.y, tx, ty, tileIdx_m, aTile[ty*TW+k],bTile[k*TW+tx]);
            }
            //printf("bx = %d, by = %d, (tx = %d, ty = %d) @ tileIdx_m = %d: c= %.2f\n",blockIdx.x, blockIdx.y, tx, ty, tileIdx_m, cvalue[tileIdx_m]);
            __syncthreads();
            c[row*m + tileCol] = cvalue[tileIdx_m];
        }

    }
  }

  if(row < n && col < m){
    for(tileIdx_m=0; tileIdx_m<NTB; tileIdx_m++){ 
        //load to C
        if((blockIdx.x % NTB) == 0){
            tileCol = tx + blockDim.x*(blockIdx.x+tileIdx_m);
            c[row*m + tileCol] = cvalue[tileIdx_m];
        }
    }
  }
    
  free(cvalue);
}


void cpu_matrixmult(float *a,float *b, float *c, int n, int m, int p) {

  int index, indexa, indexb;
  float cvalue;
  for(int col=0;col < m; col++){
    for(int row=0;row < n; row++) {
      indexb = col;
      index = row * m + col;
      cvalue = 0.;
      for (indexa = row*p; indexa < (row*p + p); indexa++, indexb+=m){ 
        cvalue += a[indexa]*b[indexb];
      }
      c[index] -= cvalue; //NOTE: This calculates the diff between CPU and GPU computations.
    }
  }
}


int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int Grid_Dim_x = 1; //Grid dimension, x
  int Grid_Dim_y = 1; //Grid dimension, y
  int Block_Dim_x = 1; //Block dimension, x
  int Block_Dim_y = 1; //Block dimension, y
  int TW = 1;
  int NTB = 1;

  int n,m,p; // matrix dimension
  float *a,*b,*c;
  float *dev_a, *dev_b, *dev_c;
  int size_a, size_b, size_c; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  //else printf("Device count = %d\n",gpucount);
  if (argc<10) {
    printf("# of inputs: %d\n", argc);
    printf("Usage: Task1GPUsp <n> <m> <p> <block dim x> <block dim y> <grid dim x> <grid dim y> <tile width> <Number of tiles>\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  m = atoi(argv[2]);
  p = atoi(argv[3]);
  

  Block_Dim_x = atoi(argv[4]); // non-Square block, # of rows
  Block_Dim_y = atoi(argv[5]); // non-Square block, # of cols
  if (Block_Dim_x * Block_Dim_y > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  //not really used in Task2 
  Grid_Dim_x = atoi(argv[6]); // non-Square grid, # of rows
  Grid_Dim_y = atoi(argv[7]); // non-Square grid, # of cols
  
  TW = atoi(argv[8]);
   
  if(Block_Dim_x != Block_Dim_y || Block_Dim_x != TW || Block_Dim_y != TW){
      printf("Error, bx, by, tw must be equal\n");
      exit(-1);
  }

  //printf("A Matrix Dimension = %dx%d\n",n,p);
  //printf("B Matrix Dimension = %dx%d\n",p,m);
  //printf("C Matrix Dimension = %dx%d\n",n,m);
  Grid_Dim_x = m/Block_Dim_x + (m % Block_Dim_x != 0);
  Grid_Dim_y = n/Block_Dim_y + (n % Block_Dim_y != 0);

  NTB = atoi(argv[9]);

  //printf("Grid_x = %d Grid_y = %d NTB = %d\n", Grid_Dim_x,Grid_Dim_y,NTB);

  dim3 Grid(Grid_Dim_x, Grid_Dim_y); //Grid structure
  dim3 Block(Block_Dim_x, Block_Dim_y); //Block structure

  size_a = n * p * sizeof(float); // number of bytes in total in arrays
  size_b = p * m * sizeof(float); // number of bytes in total in arrays
  size_c = n * m * sizeof(float); // number of bytes in total in arrays

  a = (float*) malloc(size_a); // dynamically allocated memory for arrays on host
  b = (float*) malloc(size_b);
  c = (float*) malloc(size_c); // results from GPU

  srand(12345);
  //int p = n; //Used here only to illustrate proper initialization for non-square case
   
  //printf ("a\n");
  for(i=0;i < n;i++){
    for(j=0;j < p;j++) {
      a[i * p + j] = (float) rand() / (float) RAND_MAX;
      //a[i * p + j] = (float) (i+j);
      //printf("%.2f  ", a[i * p + j]);
    }
    //printf("\n");
  }

  //printf("b\n");
  for(i=0;i < p;i++){
    for(j=0;j < m;j++) {
      b[i * m + j] = (float) rand() / (float) RAND_MAX;
      //b[i * m + j] = (float) (i+j);
      //printf("%.2f  ", b[i * m + j]);
    }
    //printf("\n");
  }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  errorcode = hipMalloc((void**)&dev_a, size_a); // allocate memory on device
    if(errorcode != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("hipMalloc error: %s\n", hipGetErrorString(errorcode));
        exit(-1);
    }
  errorcode = hipMalloc((void**)&dev_b, size_b);
    if(errorcode != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("hipMalloc error: %s\n", hipGetErrorString(errorcode));
        exit(-1);
    }
  errorcode = hipMalloc((void**)&dev_c, size_c);
    if(errorcode != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("hipMalloc error: %s\n", hipGetErrorString(errorcode));
        exit(-1);
    }

  hipMemcpy(dev_a, a , size_a ,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , size_b ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // hipEventSynchronize(start); // not needed
  size_t Ns = 2 * TW*TW * sizeof(float);
  size_t heapSize = Grid_Dim_x * Grid_Dim_y * Block_Dim_x* Block_Dim_y * NTB * sizeof(float)/4; 
  errorcode = hipDeviceSetLimit(hipLimitMallocHeapSize, heapSize);
    if(errorcode != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("cuda device heap error: %s\n", hipGetErrorString(errorcode));
        exit(-1);
    }

  matmul_tile<<<Grid,Block, Ns>>>(dev_a, dev_b, dev_c, n, m, p, TW, NTB);

    // make the host block until the device is finished with foo
    hipDeviceSynchronize();

    // check for error
    errorcode = hipGetLastError();
    if(errorcode != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(errorcode));
        exit(-1);
    }

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  hipMemcpy(c,dev_c, size_c ,hipMemcpyDeviceToHost);

  //printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time
   
  //printf("c\n");
  for(i=0;i < n;i++){
    for(j=0;j < m;j++) {
      printf("%.2f  ", c[i * m + j]);
    }
    printf("\n");
  }
  
  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)
/*
  hipEventRecord(start, 0); // use same timing

  cpu_matrixmult(a,b,c, n, m, p); // do calculation on host (NOTE: This computes the diff with GPU result.)

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------

  double error, suma, sumb, sumc, ai, bi, ci;
  suma = 0.; sumb = 0; sumc = 0;
  for(i=0;i < n*n;i++) {
    ai = (double) a[i];
    bi = (double) b[i];
    ci = (double) c[i];
    suma += ai*ai;
    sumb += bi*bi;
    sumc += ci*ci;
  }
  suma = sqrt(suma);
  sumb = sqrt(sumb);
  sumc = sqrt(sumc);
  error =  sumc/(n*suma*sumb);
  printf("Scaled error between GPU and CPU: %e\n", error);
  
*/
// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
